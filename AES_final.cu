#include "hip/hip_runtime.h"
#include <stdio.h>
#include <memory.h>
#include <stdlib.h>
#include <sys/types.h>
#include <dirent.h>
#include <iostream>
#include <fstream>
#include <vector>


#include "rijndael.h"
#include "gputimer.h"

using namespace std;

struct fileData{
    char data[8912];
    size_t size;
};

__device__ uint8_t w[4 * 4 * 15] = {0};

void err(hipError_t err)
{
	cout<<hipGetErrorString(err)<<endl;
}



__device__ static const uint32_t g_aes_rcon[] = {
    0x01000000, 0x02000000, 0x04000000, 0x08000000, 0x10000000, 0x20000000, 0x40000000, 0x80000000,
    0x1b000000, 0x36000000, 0x6c000000, 0xd8000000, 0xab000000, 0xed000000, 0x9a000000
};

/* aes sbox and invert-sbox */
__device__ static const uint8_t g_aes_sbox[256] = {
 /* 0     1     2     3     4     5     6     7     8     9     A     B     C     D     E     F  */
    0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
    0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
    0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
    0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
    0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
    0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
    0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
    0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
    0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
    0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
    0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
    0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
    0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
    0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
    0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
    0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16
};

__device__ static const uint8_t g_inv_sbox[256] = {
 /* 0     1     2     3     4     5     6     7     8     9     A     B     C     D     E     F  */
    0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38, 0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb,
    0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87, 0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb,
    0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d, 0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e,
    0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2, 0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25,
    0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92,
    0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda, 0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84,
    0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a, 0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06,
    0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02, 0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b,
    0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea, 0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73,
    0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85, 0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e,
    0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89, 0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b,
    0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20, 0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4,
    0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31, 0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f,
    0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d, 0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef,
    0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0, 0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61,
    0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26, 0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d
};

__device__ uint8_t aes_sub_sbox(uint8_t val)
{
    return g_aes_sbox[val];
}

__device__ uint32_t aes_sub_dword(uint32_t val)
{
    uint32_t tmp = 0;
   
    tmp |= ((uint32_t)aes_sub_sbox((uint8_t)((val >>  0) & 0xFF))) <<  0;
    tmp |= ((uint32_t)aes_sub_sbox((uint8_t)((val >>  8) & 0xFF))) <<  8;
    tmp |= ((uint32_t)aes_sub_sbox((uint8_t)((val >> 16) & 0xFF))) << 16;
    tmp |= ((uint32_t)aes_sub_sbox((uint8_t)((val >> 24) & 0xFF))) << 24;

    return tmp;
}

__device__ uint32_t aes_rot_dword(uint32_t val)
{
    uint32_t tmp = val;
   
    return (val >> 8) | ((tmp & 0xFF) << 24);
}

__device__ uint32_t aes_swap_dword(uint32_t val)
{
    return (((val & 0x000000FF) << 24) |
            ((val & 0x0000FF00) <<  8) |
            ((val & 0x00FF0000) >>  8) |
            ((val & 0xFF000000) >> 24) );
}

/*
 * nr: number of rounds
 * nb: number of columns comprising the state, nb = 4 dwords (16 bytes)
 * nk: number of 32-bit words comprising cipher key, nk = 4, 6, 8 (KeyLength/(4*8))
 */

__device__ void aes_key_expansion(uint8_t *key, uint8_t *round)
{
    uint32_t *w = (uint32_t *)round;
    uint32_t  t;
    int      i = 0;

    //printf("Key Expansion:\n");
    do {
        w[i] = *((uint32_t *)&key[i * 4 + 0]);
      //  printf("    %2.2d:  rs: %8.8x\n", i, aes_swap_dword(w[i]));
    } while (++i < 4);
   
    do {
        //printf("    %2.2d: ", i);
        if ((i % 4) == 0) {
            t = aes_rot_dword(w[i - 1]);
          //  printf(" rot: %8.8x", aes_swap_dword(t));
            t = aes_sub_dword(t);
            //printf(" sub: %8.8x", aes_swap_dword(t));
            //printf(" rcon: %8.8x", g_aes_rcon[i/4 - 1]);
            t = t ^ aes_swap_dword(g_aes_rcon[i/4 - 1]);
            //printf(" xor: %8.8x", t);
        } else if (4 > 6 && (i % 4) == 4) {
            t = aes_sub_dword(w[i - 1]);
            //printf(" sub: %8.8x", aes_swap_dword(t));
        } else {
            t = w[i - 1];
            //printf(" equ: %8.8x", aes_swap_dword(t));
        }
        w[i] = w[i - 4] ^ t;
        //printf(" rs: %8.8x\n", aes_swap_dword(w[i]));
    } while (++i < 4 * (10 + 1));
   
    /* key can be discarded (or zeroed) from memory */
}

__device__ void aes_add_round_key(uint8_t *state,
                       uint8_t *round, int nr)
{
    uint32_t *w = (uint32_t *)round;
    uint32_t *s = (uint32_t *)state;
    int i;
   
    for (i = 0; i < 4; i++) {
        s[i] ^= w[nr * 4 + i];
    }
}

__device__ void aes_sub_bytes(uint8_t *state)
{
    int i, j;
   
    for (i = 0; i < 4; i++) {
        for (j = 0; j < 4; j++) {
            state[i * 4 + j] = aes_sub_sbox(state[i * 4 + j]);
        }
    }
}

__device__ void aes_shift_rows(uint8_t *state)
{
    uint8_t *s = (uint8_t *)state;
    int i, j, r;
   
    for (i = 1; i < 4; i++) {
        for (j = 0; j < i; j++) {
            uint8_t tmp = s[i];
            for (r = 0; r < 4; r++) {
                s[i + r * 4] = s[i + (r + 1) * 4];
            }
            s[i + (4 - 1) * 4] = tmp;
        }
    }
}

__device__ uint8_t aes_xtime(uint8_t x)
{
    return ((x << 1) ^ (((x >> 7) & 1) * 0x1b));
}

__device__ uint8_t aes_xtimes(uint8_t x, int ts)
{
    while (ts-- > 0) {
        x = aes_xtime(x);
    }
   
    return x;
}

__device__ uint8_t aes_mul(uint8_t x, uint8_t y)
{
    /*
     * encrypt: y has only 2 bits: can be 1, 2 or 3
     * decrypt: y could be any value of 9, b, d, or e
     */
   
    return ((((y >> 0) & 1) * aes_xtimes(x, 0)) ^
            (((y >> 1) & 1) * aes_xtimes(x, 1)) ^
            (((y >> 2) & 1) * aes_xtimes(x, 2)) ^
            (((y >> 3) & 1) * aes_xtimes(x, 3)) ^
            (((y >> 4) & 1) * aes_xtimes(x, 4)) ^
            (((y >> 5) & 1) * aes_xtimes(x, 5)) ^
            (((y >> 6) & 1) * aes_xtimes(x, 6)) ^
            (((y >> 7) & 1) * aes_xtimes(x, 7)) );
}

__device__ void aes_mix_columns(uint8_t *state)
{
    uint8_t y[16] = { 2, 3, 1, 1,  1, 2, 3, 1,  1, 1, 2, 3,  3, 1, 1, 2};
    uint8_t s[4];
    int i, j, r;
   
    for (i = 0; i < 4; i++) {
        for (r = 0; r < 4; r++) {
            s[r] = 0;
            for (j = 0; j < 4; j++) {
                s[r] = s[r] ^ aes_mul(state[i * 4 + j], y[r * 4 + j]);
            }
        }
        for (r = 0; r < 4; r++) {
            state[i * 4 + r] = s[r];
        }
    }
}


void aes_dump(char *msg, uint8_t *data, int len)
{
    int i;
   
    printf("%8.8s: ", msg);
    for (i = 0; i < len; i++) {
        printf(" %2.2x", data[i]);
    }
    printf("\n");
}

__global__ void aes_encrypt(uint8_t *data, uint8_t *res,int len, uint8_t *key)
{
    __shared__ uint8_t s[4 * 4]; /* state */
   
    int nr, i, j;

    

   
    /* key expansion */
    if((blockIdx.x==0)&&(threadIdx.x==0))
    aes_key_expansion(key, w);
    __syncthreads();
   
    /* start data cypher loop over input buffer */
    for (i = 0; i < len; i += 4 * 4) {


        /* init state from user buffer (plaintext) */
        for (j = 0; j < 4 * 4; j++)
            s[j] = data[(blockIdx.x*4096)+i + j];
       
        /* start AES cypher loop over all AES rounds */
        for (nr = 0; nr <= 10; nr++) {
           
           
            if (nr > 0) {
               
                /* do SubBytes */
                aes_sub_bytes(s);
               
                /* do ShiftRows */
                aes_shift_rows(s);
               
                if (nr < 10) {
                    /* do MixColumns */
                    aes_mix_columns(s);
                }
            }
           
            /* do AddRoundKey */
            aes_add_round_key(s, w, nr);
        }
       
        /* save state (cypher) to user buffer */
        for (j = 0; j < 4 * 4; j++)
            res[(blockIdx.x*4096)+i + j] = s[j];
    }
   
}


__device__ void inv_shift_rows(uint8_t *state)
{
    uint8_t *s = (uint8_t *)state;
    int i, j, r;
   
    for (i = 1; i < 4; i++) {
        for (j = 0; j < 4 - i; j++) {
            uint8_t tmp = s[i];
            for (r = 0; r < 4; r++) {
                s[i + r * 4] = s[i + (r + 1) * 4];
            }
            s[i + (4 - 1) * 4] = tmp;
        }
    }
}

__device__ uint8_t inv_sub_sbox(uint8_t val)
{
    return g_inv_sbox[val];
}


__device__ void inv_sub_bytes(uint8_t *state)
{
    int i, j;
   
    for (i = 0; i < 4; i++) {
        for (j = 0; j < 4; j++) {
            state[i * 4 + j] = inv_sub_sbox(state[i * 4 + j]);
        }
    }
}

__device__ void inv_mix_columns(uint8_t *state)
{
    uint8_t y[16] = { 0x0e, 0x0b, 0x0d, 0x09,  0x09, 0x0e, 0x0b, 0x0d,
                      0x0d, 0x09, 0x0e, 0x0b,  0x0b, 0x0d, 0x09, 0x0e};
    uint8_t s[4];
    int i, j, r;
   
    for (i = 0; i < 4; i++) {
        for (r = 0; r < 4; r++) {
            s[r] = 0;
            for (j = 0; j < 4; j++) {
                s[r] = s[r] ^ aes_mul(state[i * 4 + j], y[r * 4 + j]);
            }
        }
        for (r = 0; r < 4; r++) {
            state[i * 4 + r] = s[r];
        }
    }
}

__global__ void aes_decrypt(uint8_t *data, uint8_t *res,int len, uint8_t *key)
{
    
    __shared__ uint8_t s[4 * 4]; /* state */
   
    int nr, i, j;
   
    /* key expansion */
    if((blockIdx.x==0)&&(threadIdx.x==0))
    aes_key_expansion(key, w);
    __syncthreads();
   

   
    /* start data cypher loop over input buffer */
    for (i = 0; i < len; i += 4 * 4) {
       
       
        /* init state from user buffer (cyphertext) */
        for (j = 0; j < 4 * 4; j++)
            s[j] = data[(blockIdx.x*4096)+i + j];
       
        /* start AES cypher loop over all AES rounds */
        for (nr = 10; nr >= 0; nr--) {
           
 
            /* do AddRoundKey */
            aes_add_round_key(s, w, nr);
            //aes_dump("  round", &w[nr * 4 * 4], 4 * 4);


            if (nr > 0) {

                if (nr < 10) {
                    /* do MixColumns */
                    inv_mix_columns(s);
                }

                /* do ShiftRows */
                inv_shift_rows(s);

                /* do SubBytes */
                inv_sub_bytes(s);
            }
           
        }
       
        /* save state (cypher) to user buffer */
        for (j = 0; j < 4 * 4; j++)
            res[(blockIdx.x*4096)+i + j] = s[j];
    }
   
}



void read_directory(const string& name, vector<string>& v)
{
    DIR* dirp = opendir(name.c_str());
    struct dirent * dp;
    while ((dp = readdir(dirp)) != NULL) {
        if(dp->d_name[0]=='.')
            continue;
        v.push_back(dp->d_name);
    }
    closedir(dirp);
}


string get_file_content(const char* filename)
{
    ifstream in(filename, std::ios::in | std::ios::binary);
    if (in)
    {
        std::string contents;
        in.seekg(0, std::ios::end);
        contents.resize(in.tellg());
        in.seekg(0, std::ios::beg);
        in.read(&contents[0], contents.size());
        in.close();
        return(contents);
    }
    throw(errno);
}


void aes_cypher_128_test()
{
    //unsigned char h_buffer[8192],h_cipherText[8192],h_res[8192];
    //fileData *m_buffer,*m_cipherText,*m_res;
    unsigned char *d_buffer,*d_cipherText,*d_res;
    uint8_t *d_key;
    GpuTimer allocTimer,memcpyTimer,encrypt_executionTimer,decrypt_executionTimer;
    /*FILE *fp;
    int i;
    fp = fopen(t,"rb");
    if(fp==NULL)
        {
            printf("FIle open failed\n");
            exit(1);
        }
    int byteRead = fread(h_buffer,1,8196,fp);
    printf("byteRead:%d\n",byteRead);*/


  // ------------------------------------------------------File Handling------------------------------------------------------------
    string path = "/home/aniket/out";
    vector<string> fileList;
    vector<string> fileContent;

    read_directory(path,fileList);
    for(unsigned int i=0;i<fileList.size();i++)
        fileContent.push_back(get_file_content((path+"/"+fileList[i]).c_str()));

   //-------------------------------------------------------------------------------------------------------------------------------


    uint8_t h_key[] = { 0x2b, 0x7e, 0x15, 0x16, 0x28, 0xae, 0xd2, 0xa6,
                      0xab, 0xf7, 0x15, 0x88, 0x09, 0xcf, 0x4f, 0x3c };
    



    allocTimer.Start();
    err(hipMallocManaged((void**)&d_buffer,sizeof(unsigned char)*588*4096));
    err(hipMallocManaged((void**)&d_cipherText,sizeof(unsigned char)*588*4096));
    err(hipMalloc((void**)&d_key,sizeof(uint8_t)*16));
    err(hipMallocManaged((void**)&d_res,sizeof(unsigned char)*588*4096));
    allocTimer.Stop();
    err(hipMemcpy(d_key, h_key, 16, hipMemcpyHostToDevice));
    
    memcpyTimer.Start();
    for(int i = 0; i < 588; i++)
    {
        strcpy((char *)(d_buffer+i*4096),fileContent[i].c_str());
    }
    memcpyTimer.Stop();

    encrypt_executionTimer.Start();
    aes_encrypt<<<fileList.size(),1>>>(d_buffer, d_cipherText, 4096, d_key);
    encrypt_executionTimer.Stop();
    hipDeviceSynchronize();
    //hipMemcpy(m_cipherText, d_cipherText, sizeof(char)*byteRead, hipMemcpyDeviceToHost);

   

   /* for(int i = 0; i<4032;i++)
        printf("%c", d_cipherText[i]);
    printf("\n");//*/
   // hipMemcpy(d_cipherText, h_cipherText, sizeof(char)*byteRead, hipMemcpyHostToDevice);
    
    //for(i = 0;i<byteRead;i+=16)
    decrypt_executionTimer.Start();
    aes_decrypt<<<fileList.size(),1>>>(d_cipherText, d_res,4096, d_key);
    decrypt_executionTimer.Stop();
    hipDeviceSynchronize();
    //hipMemcpy((void*)h_res, d_res, sizeof(char)*byteRead, hipMemcpyDeviceToHost);



   

   cout<<"Elapsed Times:\n";
   cout<<"Allocation Time:      "<<allocTimer.Elapsed()<<"ms"<<endl;
   cout<<"Copy Time:            "<<memcpyTimer.Elapsed()<<"ms"<<endl;
   cout<<"Encryption Time:      "<<encrypt_executionTimer.Elapsed()<<"ms"<<endl;
   cout<<"Decryption Time:      "<<decrypt_executionTimer.Elapsed()<<"ms"<<endl;
   /*for(int i = 587*4096; i < 588*4096;i++)
   		printf("%c", d_res[i]);
    printf("\n");//*/
}

int main(int argc,char** argv)
{
    aes_cypher_128_test();
    return 0;
}